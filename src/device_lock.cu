#include "hip/hip_runtime.h"
#include "device_lock.hh"

DeviceLock::DeviceLock()
{
    hipMalloc(&lock_, sizeof(int));
    hipMemset(lock_, 0, sizeof(int));
}

DeviceLock::~DeviceLock()
{
    hipFree(lock_);
}

__device__
void DeviceLock::lock()
{
    while (atomicCAS(lock_, 0, 1) != 0)
        continue;
}

__device__
void DeviceLock::unlock()
{
    atomicExch(lock_, 0);
}

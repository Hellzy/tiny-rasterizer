#include "hip/hip_runtime.h"
#include "gpu_operations.hh"

template <typename T, unsigned dim2>
__device__ void device_mat_mult(T m1[9], T m2[dim2], T out[dim2])
{
    int out_off = 0;

    for (unsigned i = 0; i < 3; ++i)
    {
        for (unsigned j = 0; j < dim2 / 3; j++)
        {
            for (unsigned k = 0; k < 3; ++k)
                out[out_off] += m1[i * 3 + k] * m2[k * 3 / dim2 + j];

            ++out_off;
        }
    }
}

__device__ void cam_project_point(const cam_t& cam, point_t& p)
{
    double rot_mat[] =
    {
        cam.dir_x.x, cam.dir_x.y, cam.dir_x.z,
        cam.dir_y.x, cam.dir_y.y, cam.dir_y.z,
        cam.dir_z.x, cam.dir_z.y, cam.dir_z.z
    };

    p.x -= cam.pos.x;
    p.y -= cam.pos.y;
    p.z -= cam.pos.z;

    double trans_mat[] = { p.x, p.y, p.z };
    double out_mat[3] = { 0 };

    device_mat_mult<double, 3>(rot_mat, trans_mat, out_mat);

    p.x = out_mat[0];
    p.y = out_mat[1];
    p.z = out_mat[2];
}

__global__ void cuda_project_points(point_t* points, size_t point_nb, cam_t cam, size_t screen_w, size_t screen_h)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < point_nb)
    {
        constexpr double ncp = 1;
        point_t p = points[idx];

        cam_project_point(cam, p);

        /* Screen projection */
        p.x = ncp * (screen_w / 2.0) * p.x / -p.z + screen_w / 2.0;
        p.y = ncp * (screen_h / 2.0) * p.y / -p.z + screen_h / 2.0;
        p.z = -p.z;

        /* NDC projection */
        double l = 0;
        double r = screen_w;
        double b = 0;
        double t = screen_h;

        p.x = 2 * p.x / (r - l) - (r + l) / (r - l);
        p.y = 2 * p.y / (t - b) - (t + b) / (t - b);

        /* Raster projection */
        p.x = (p.x + 1) / 2 * screen_w;
        p.y = (1 - p.y) / 2 * screen_h;

        points[idx] = p;
    }
}

void projection_kernel(point_t* points, size_t point_nb, const cam_t& cam, size_t screen_w, size_t screen_h)
{
    point_t* points_d;

    hipMalloc(&points_d, sizeof(point_t) * point_nb);
    hipMemcpy(points_d, points, sizeof(point_t) * point_nb,
            hipMemcpyHostToDevice);
    cuda_project_points<<<point_nb / 1024 + 1, 1024>>>(points_d, point_nb, cam, screen_w, screen_h);
    hipMemcpy(points, points_d, sizeof(point_t) * point_nb,
            hipMemcpyDeviceToHost);
    hipFree(points_d);
}
